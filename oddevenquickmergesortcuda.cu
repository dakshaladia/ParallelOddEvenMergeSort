#include "hip/hip_runtime.h"
%%cu
#include "hip/hip_runtime.h"
#include ""
#include<stdio.h>
#include<stdlib.h>
#define CUDA_ERROR_CHECK

__device__ void prez(int i,int *d,int s,int p,int *r)
{   
    int index=i*(s/p);
    int index2=(i+1)*(s/p);
    int c11=0,c21=0,c12=s/p-1,c22=s/p-1;

    for(int k=0;k<s/p;k++)
    {
        if(d[index+c11]<=d[index2+c21])
        {
            r[k]=d[index+c11];
            c11++;
        }
        else
        {
            r[k]=d[index2+c21];
            c21++;
        }
    }
    __syncthreads();
    for(int k=0;k<s/p;k++)
    {
        if(d[index2+c22]<d[index2+c12])
        {
            r[(2*(s/p))-1-k]=d[index2+c22];
            c22--;
        }
        else
        {
            r[(2*(s/p))-1-k]=d[index+c12];
            c12--;
        }
    }
    __syncthreads();
    
    int rt=0;
        for(int y=0;y<(s/p);y++)
        {
            d[index+y]=r[rt++];
        }
        for(int y=0;y<(s/p);y++)
        {
            d[index2+y]=r[rt++];
        }
    __syncthreads();
}

__global__ void pcm (int *d,int s,int p,int x)
{   
    int i=blockIdx.x;
    int index=i*blockDim.x;
    int len=((s/p)*2);
    int tid=threadIdx.x;
    int size=sizeof(int)*len;

    if(x==0 && ((i*2+1)<p))
    { 
        int len=((s/p)*2);
        int size=sizeof(int)*len;
        int *r=(int *)malloc(sizeof(int)*len);
        prez(i*2,d,s,p,r);
        __syncthreads();
    }
    else if(x==1 && ((i*2+2)<p))
    {
        int len=((s/p)*2);
        int size=sizeof(int)*len;
        int *r=(int *)malloc(sizeof(int)*len);
        prez(i*2+1,d,s,p,r);
        __syncthreads();
    }
}  
   
__device__ void quicksort(int *number,int first,int last)
{
   int i, j, pivot, temp;
   if(first<last)
   {
        pivot=first;
        i=first;
        j=last;
    while(i<j)
    {
    while(number[i]<=number[pivot]&&i<last)
        i++;
        while(number[j]>number[pivot])
        j--;
        if(i<j)
        {
            temp=number[i];
            number[i]=number[j];
            number[j]=temp;
        }
    }

    temp=number[pivot];
    number[pivot]=number[j];
    number[j]=temp;
    quicksort(number,first,j-1);
    quicksort(number,j+1,last);
    }
}

__global__ void qsort(int *d,int s,int p,int *fg)
{
    int i=blockIdx.x;
    int index=i*blockDim.x;
    
    for(int j=0;j<(s/p);j++)
    {
       fg[index+j]=d[index+j];
    }
    
    quicksort(fg,index,index+(s/p)-1);
     
    for(int j=0;j<(s/p);j++)
    {
       d[index+j]=fg[index+j]; 
    }
  
    for(int e=0;e<s/p;e++)
    {
        
        printf("%d ",d[e]);
           
    }
    printf("\n");*/
}

int main()
{   
    int p=2,s=12;
    hipEvent_t start, stop;     		
	float elapsed_time_ms;  
    int *d_a,*d_r,*d_f;
    int fg[s]={0};
    int d[12]={12,11,10,9,8,7,6,5,4,3,2,1};
    printf("Original Array:\n");
    for(int i=0;i<p*(s/p);i++)
    {
        printf("%d ",d[i]);
    }
    printf("\n\n\n");
    int size=(sizeof(int)*s);
    hipMalloc((void**)&d_a,size);
    hipMalloc((void**)&d_f,size);
    int lenn=s/p;
    hipEventCreate(&start);     		
	hipEventCreate(&stop);
    hipEventRecord(start, 0);
    hipMemcpy(d_a,&d,size,hipMemcpyHostToDevice);
    hipMemcpy(d_f,&fg,size,hipMemcpyHostToDevice);
    qsort<<<p,lenn>>>(d_a,s,p,d_f);
    hipDeviceSynchronize();
    hipError_t error =hipGetLastError();
    if (error != hipSuccess)
    {
        printf("CUDA Error1: %s\n", hipGetErrorString(error));
    }

    hipMemcpy(&fg,d_f,size,hipMemcpyDeviceToHost);
    hipMemcpy(&d,d_a,size,hipMemcpyDeviceToHost);
    hipMemcpy(d_a,&d,size,hipMemcpyHostToDevice);
    printf("QQSorted Array\n");
    for(int e=0;e<s;e++)
    {
        printf("%d ",d[e]);
    }
    printf("\n");
    for(int pp=0;pp<p;pp++)
        pcm<<<p/2,1>>>(d_a,s,p,pp%2);
 
    hipDeviceSynchronize();
    error =hipGetLastError();
    if (error != hipSuccess)
    {
        printf("CUDA Error2: %s\n", hipGetErrorString(error));
    }
    hipMemcpy(&d,d_a,size,hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
 
    printf("Sorted Array: \n");
    for(int e=0;e<s;e++)
    {
        printf("%d ",d[e]);
    }
    printf("\n");
    hipEventRecord(stop, 0);     	
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time_ms, start, stop );
    printf("\nTime taken for the entire computation: %f ms.\n", elapsed_time_ms);
    hipFree(d_a);
}
